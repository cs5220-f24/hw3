
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <math.h>

// function to add the elements of two arrays
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements
  std::vector<float> x(N);
  std::vector<float> y(N);

  // initialize x and y arrays on the host
  for (auto& xi : x) xi = 1.0f;
  for (auto& yi : y) yi = 2.0f;

  // Run kernel on 1M elements on the CPU
  add(N, x.data(), y.data());

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (auto yi : y)
    maxError = fmax(maxError, fabs(yi-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  return 0;
}
